#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include "../include/rle.h"

__global__ void prefix_sum(const int *input, int *output, int n, bool exclusive) {
    __shared__ int temp[1024];
    int tid = threadIdx.x;

    temp[tid] = (tid < n) ? input[tid] : 0;
    __syncthreads();

    for (int offset = 1; offset < n; offset *= 2) {
        int t = temp[tid];
        if (tid >= offset)
            t += temp[tid - offset];
        __syncthreads();
        temp[tid] = t;
        __syncthreads();
    }

    if (tid < n) {
        if (exclusive)
            output[tid] = (tid == 0) ? 0 : temp[tid - 1];
        else
            output[tid] = temp[tid];
    }
}

__global__ void mark_series(const uint8_t *input, int *flags, size_t n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= n) return;

    if (i == 0 || input[i] != input[i - 1])
        flags[i] = 1;
    else
        flags[i] = 0;
}

__global__ void write_compressed(const uint8_t *input, size_t n,
                          const int *flags, const int *positions,
                          uint8_t *output) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= n) return;

    if (flags[i] == 1) {
        int start = i;
        uint8_t byte = input[i];
        int len = 1;
        while (start + len < n && input[start + len] == byte)
            len++;

        int out_pos = positions[i] * 2;
        output[out_pos] = byte;
        output[out_pos + 1] = (uint8_t)(len);
    }
}

__global__ void write_decompressed(const uint8_t *input,
                                   const int *positions,
                                   uint8_t *output, int num_pairs) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= num_pairs) return;

    uint8_t value = input[2 * i];
    int count = input[2 * i + 1];
    int start = positions[i];

    for (int j = 0; j < count; ++j) {
        output[start + j] = value;
    }
}

int rle_compress_cuda(const uint8_t *input, size_t input_size,
                      uint8_t *output, size_t *output_size) {
    if (!input || !output || !output_size || input_size == 0)
        return -1;

    uint8_t *d_input, *d_output;
    int *d_flags, *d_positions;
    size_t max_output_size = input_size * 2;

    hipMalloc((void**)&d_input, input_size);
    hipMalloc((void**)&d_output, max_output_size);
    hipMalloc((void**)&d_flags, input_size * sizeof(int));
    hipMalloc((void**)&d_positions, input_size * sizeof(int));
    hipMemcpy(d_input, input, input_size, hipMemcpyHostToDevice);

    int threads = 256;
    int blocks = (input_size + threads - 1) / threads;

    mark_series<<<blocks, threads>>>(d_input, d_flags, input_size);
    hipDeviceSynchronize();

    prefix_sum<<<1, 1024>>>(d_flags, d_positions, input_size, false);
    hipDeviceSynchronize();

    write_compressed<<<blocks, threads>>>(d_input, input_size, d_flags, d_positions, d_output);
    hipDeviceSynchronize();

    int last_pos = 0;
    hipMemcpy(&last_pos, &d_positions[input_size - 1], sizeof(int), hipMemcpyDeviceToHost);
    int num_series = last_pos + 1;

    hipMemcpy(output, d_output, num_series * 2, hipMemcpyDeviceToHost);
    *output_size = num_series * 2;

    hipFree(d_input);
    hipFree(d_output);
    hipFree(d_flags);
    hipFree(d_positions);

    return 0;
}

int rle_decompress_cuda(const uint8_t *input, size_t input_size,
                        uint8_t *output, size_t *output_size) {
    if (!input || !output || !output_size || input_size % 2 != 0)
        return -1;

    int num_pairs = input_size / 2;

    uint8_t *d_input, *d_output;
    int *d_counts, *d_positions;

    hipMalloc((void**)&d_input, input_size);
    hipMemcpy(d_input, input, input_size, hipMemcpyHostToDevice);
    hipMalloc((void**)&d_counts, num_pairs * sizeof(int));
    hipMalloc((void**)&d_positions, num_pairs * sizeof(int));

    // Извлекаем длины серий
    int *h_counts = (int*)malloc(num_pairs * sizeof(int));
    for (int i = 0; i < num_pairs; ++i)
        h_counts[i] = input[2 * i + 1];
    hipMemcpy(d_counts, h_counts, num_pairs * sizeof(int), hipMemcpyHostToDevice);
    free(h_counts);

    // Префикс-сумма по длинам
    prefix_sum<<<1, 1024>>>(d_counts, d_positions, num_pairs, true);
    hipDeviceSynchronize();

    int last_pos = 0, last_len = input[input_size - 1];
    hipMemcpy(&last_pos, &d_positions[num_pairs - 1], sizeof(int), hipMemcpyDeviceToHost);
    *output_size = last_pos + last_len;

    hipMalloc((void**)&d_output, *output_size);

    int threads = 256;
    int blocks = (num_pairs + threads - 1) / threads;

    write_decompressed<<<blocks, threads>>>(d_input, d_positions, d_output, num_pairs);
    hipDeviceSynchronize();

    hipMemcpy(output, d_output, *output_size, hipMemcpyDeviceToHost);

    hipFree(d_input);
    hipFree(d_output);
    hipFree(d_counts);
    hipFree(d_positions);

    return 0;
}
